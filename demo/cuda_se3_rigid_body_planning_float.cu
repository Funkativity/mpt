#include "hip/hip_runtime.h"
// Software License Agreement (BSD-3-Clause)
//
// Copyright 2018 The University of North Carolina at Chapel Hill
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//
// 1. Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above
//    copyright notice, this list of conditions and the following
//    disclaimer in the documentation and/or other materials provided
//    with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
// COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
// INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
// STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED
// OF THE POSSIBILITY OF SUCH DAMAGE.

//! @author Jeff Ichnowski

#include "cuda_se3_rigid_body_scenario_float.hpp"
#include "scenario_config.hpp"
#include "proc_info.hpp"
#include <mpt/pprm.hpp>
#include <mpt/prrt.hpp>
#include <mpt/prrt_star.hpp>
#include <nigh/gnat.hpp>
#include <getopt.h>
#include <iomanip>
#include <chrono>

// To compile:
// cd ~/projects/mpt/demo ; clang++-9 -std=c++17 -x cuda --cuda-path=/usr/local/cuda-9.2/ -I../src -I/usr/include/assimp -I../../nigh/src -I/usr/include/fcl -I/usr/local/include/eigen3 -o cuda_se3_rigid_body_planning cuda_se3_rigid_body_planning.cu -lfcl -lassimp --cuda-gpu-arch=sm_61 -lccd -lpthread
// clang++-9 -std=c++17 -x cuda --cuda-path=/usr/local/cuda-9.2/ -I../src -I/usr/include/assimp -I../../nigh/src -I/usr/include/fcl -I/usr/local/include/eigen3 -o cuda_se3_rigid_body_planning cuda_se3_rigid_body_planning.cu -lfcl -lassimp --cuda-gpu-arch=sm_61 -lccd -lpthread -L/usr/local/cuda/lib64 -lcudart -O3 -D MT=0 -D NN_TYPE=KDTreeBatch -D SCALAR_TYPE=float
// ./cuda_se3_rigid_body_planning -S -a prm ../../omplapp-1.4.2-Source/resources/3D/Easy.cfg 
enum PlanningAlgorithm {
    kRRTStarAlgorithm,
    kRRTAlgorithm,
    kPRMAlgorithm,
};

struct Options {
    int solveTimeMillis_{-1};
    int nodeCount_{-1};
    bool terminateWhenSolved_{false};
    PlanningAlgorithm algorithm_{kRRTStarAlgorithm};

    enum {
        kScalarFloat,
        kScalarDouble,
        // long double isn't fully supported by FCL
        //kScalarLongDouble
    } scalarType_{kScalarDouble};

    std::string path_;
    std::string configFile_;

    Options(int argc, char *argv[]) {
        static struct option options[] = {
            { "solve-time", required_argument, 0, 't' },
            { "solved", no_argument, 0, 'S' },
            { "nodes", required_argument, 0, 'n' },
            { "algorithm", required_argument, 0, 'a' },
            { "scalar", required_argument, 0, 's' },
            { nullptr, 0, nullptr, 0}
        };

        for (int c, optInd ; -1 != (c=getopt_long(argc, argv, "t:s:n:a:S", options, &optInd)) ; ) {
            std::size_t pos;
            std::string arg;

            switch (c) {
            case 't':
                arg = optarg;
                solveTimeMillis_ = std::stoi(arg, &pos);
                if (pos != arg.length() || solveTimeMillis_ < 0)
                    throw std::invalid_argument("invalid solve time: " + arg);
                break;
            case 'n':
                arg = optarg;
                nodeCount_ = std::stoi(arg, &pos);
                if (pos != arg.length() || nodeCount_ < 0)
                    throw std::invalid_argument("invalid node count: " + arg);
                break;
            case 'S':
                terminateWhenSolved_ = true;
                break;
            case 'a':
                if (std::strcmp("rrtstar", optarg) == 0) {
                    algorithm_ = kRRTStarAlgorithm;
                } else if (std::strcmp("rrt", optarg) == 0) {
                    algorithm_ = kRRTAlgorithm;
                } else if (std::strcmp("prm", optarg) == 0) {
                    algorithm_ = kPRMAlgorithm;
                } else {
                    throw std::invalid_argument("expected algorithm to be 'rrtstar' or 'rrt'");
                }
                break;
            case 's':
                if (std::strcmp("float", optarg) == 0) {
                    scalarType_ = kScalarFloat;
                } else if (std::strcmp("double", optarg) == 0) {
                    scalarType_ = kScalarDouble;
                // } else if (std::strcmp("long_double", optarg) == 0) {
                //     scalarType_ = kScalarLongDouble;
                } else {
                    throw std::invalid_argument("expected scalar type of 'float' or 'double'"); // , or 'long_double'");
                }
                break;
            default:
                std::cerr << "usage: " << argv[0] << " [options] config-file.cfg\n"
                    "Options:\n"
                    "  -t --solve-time=TIME   Specify the time in milliseconds to spend solving\n"
                    "  -S --solved            Run until solved\n"
                    "  -n --nodes=N           Run until planner has generated N\n"
                    "  -a --algorithm=ALG     Run the planning algorithm (rrt or rrtstar)\n"
                    "  -t TIME\n"
                          << std::flush;
                throw std::invalid_argument("unrecognized option");
            }
        }

        if (optind+1 != argc)
            throw std::invalid_argument("expected configuration file");

        configFile_ = argv[optind];

        std::size_t lastSlash = configFile_.find_last_of("\\/");
        path_ = (lastSlash == std::string::npos) ? "" : configFile_.substr(0, lastSlash+1);
    }
};

template <typename Scenario, typename Algorithm, typename Config, typename Scalar>
void runPlanner(
    const Options& options,
    const mpt_demo::ScenarioConfig<>& config,
    const std::string& envMesh, const std::vector<std::string>& robotMeshes,
    const Config& qStart,
    const Config& qGoal,
    const Eigen::Matrix<Scalar, 3, 1>& volumeMin,
    const Eigen::Matrix<Scalar, 3, 1>& volumeMax)
{
    using namespace unc::robotics::nigh;
    using namespace unc::robotics::mpt;
    using namespace std::literals;

    MPT_LOG(INFO) << "Algorithm: " << log::type_name<Algorithm>();
    using Clock = std::chrono::steady_clock;

    // call the SE3RigidBodyScenario constructor defined in the header
    Scenario scenario(envMesh, robotMeshes, qGoal, volumeMin, volumeMax, 0.01);

    Planner<Scenario, Algorithm> planner(scenario);
    // planner.addGoal(qGoal);
    planner.addStart(qStart);

    // TODO: only do this for Algorithms that have setRange
//     if (config.hasProp("planner", "rrt.range")) {
//         Scalar range;
//         config.load(range, "planner", "rrt.range");
//         MPT_LOG(INFO) << "setting range: " << range;
//         planner.setRange(range);
//     }

    Clock::time_point start;
    if (options.solveTimeMillis_ > 0) {
        MPT_LOG(INFO) << "solving for " << options.solveTimeMillis_ << " ms";
        start = Clock::now();
        planner.solveFor(std::chrono::milliseconds(options.solveTimeMillis_));
    } else if (options.nodeCount_ > 0) {
        MPT_LOG(INFO) << "solving to node count " << options.nodeCount_;
        start = Clock::now();
        planner.solve([&, count = (std::size_t)options.nodeCount_] () { return planner.size() >= count; });
    } else if (options.terminateWhenSolved_) {
        MPT_LOG(INFO) << "solving until solution";
        start = Clock::now();
        // solve takes in a function that determines when it stops, this is a lambda that only stops when the planner is finished
        planner.solve([&] () { return planner.solved(); });  

    }
    Clock::duration elapsed = Clock::now() - start;
    // TODO: planner.logStats();

    MPT_LOG(INFO) << "planner generated " << planner.size() << " states";
    MPT_LOG(INFO) << "solve time " << std::chrono::duration<double>(elapsed).count() << " seconds";

    planner.printStats();

    if (planner.solved()) {
        std::vector<Config> path = planner.solution();

        Scalar cost = 0;
        std::cout << "Path: " <<std::endl;
        auto curr = path.begin();
        if (curr != path.end()) {
            for (auto prev = curr ; ++curr != path.end() ; prev = curr){
                std::cout << *curr << std::endl;
                cost += scenario.space().distance(*prev, *curr);
            }
        }
        MPT_LOG(INFO) << "path cost " << cost;

        if (false) {
            for (const Config& q : path) {
                std::cout << std::setprecision(std::numeric_limits<Scalar>::digits10 + 1)
                          << std::get<1>(q)[0] << ' '
                          << std::get<1>(q)[1] << ' '
                          << std::get<1>(q)[2] << ' '
                          << std::get<0>(q).coeffs()[0] << ' '
                          << std::get<0>(q).coeffs()[1] << ' '
                          << std::get<0>(q).coeffs()[2] << ' '
                          << std::get<0>(q).coeffs()[3] << '\n';
            }
        }
    }

    std::cout << mpt_demo::ProcInfo() << std::flush;
}

template <typename Scalar>
void solve(const Options& options, const mpt_demo::ScenarioConfig<>& config) {
    using namespace unc::robotics::mpt;
    using namespace unc::robotics::nigh;

    using Scenario = mpt_demo::SE3RigidBodyScenario<Scalar>;
    using Space = typename Scenario::Space;
    using Config = typename Space::Type;

    Config qGoal;
    Config qStart;

    config.load(qGoal, "problem", "goal");
    config.load(qStart, "problem", "start");

    MPT_LOG(INFO) << "start: " << qStart;
    MPT_LOG(INFO) << "goal: " << qGoal;

    std::string envMesh;
    config.load(envMesh, "problem", "world");
    envMesh = options.path_ + envMesh;

    std::string robotMesh;
    config.load(robotMesh, "problem", "robot");
    robotMesh = options.path_ + robotMesh;

    Eigen::Matrix<Scalar, 3, 1> volumeMin;
    Eigen::Matrix<Scalar, 3, 1> volumeMax;
    config.load(volumeMin, "problem", "volume.min");
    config.load(volumeMax, "problem", "volume.max");

    using NN = NN_TYPE<>;

// #if MT
    // using Threads = hardware_concurrency;
// #else
    using Threads = single_threaded;
// #endif

    static constexpr bool reportStats = true;

    std::vector<std::string> robotMeshes{{robotMesh}};
    if (options.algorithm_ == kRRTStarAlgorithm) {
        using Algorithm = PRRTStar<report_stats<reportStats>, NN, Threads>;
        runPlanner<Scenario, Algorithm>(options, config, envMesh, robotMeshes, qStart, qGoal, volumeMin, volumeMax);
    } else if (options.algorithm_ == kRRTAlgorithm) {
        using Algorithm = PRRT<report_stats<reportStats>, NN, Threads>;
        runPlanner<Scenario, Algorithm>(options, config, envMesh, robotMeshes, qStart, qGoal, volumeMin, volumeMax);
    } else if (options.algorithm_ == kPRMAlgorithm) {
        using Algorithm = PPRM<report_stats<reportStats>, NN, Threads>;
        runPlanner<Scenario, Algorithm>(options, config, envMesh, robotMeshes, qStart, qGoal, volumeMin, volumeMax);
    }
}


int main(int argc, char *argv[]) {
    using namespace mpt_demo;

    try {
        Options options(argc, argv);

        ScenarioConfig<> config(options.configFile_);

        using Scalar = SCALAR_TYPE;

        solve<Scalar>(options, config);

        // switch (options.scalarType_) {
        // case Options::kScalarFloat:
        //     solve<float>(options, config);
        //     break;
        // case Options::kScalarDouble:
        //     solve<double>(options, config);
        //     break;
        // // case Options::kScalarLongDouble:
        // //     solve<long double>(options, config);
        // //     break;
        // }
        return 0;
    } catch (const std::exception& ex) {
        MPT_LOG(FATAL) << "terminated with exception: " << ex.what();
        return 1;
    }
}
